#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/swish_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

template <typename T>
T *CreateSwishPluginDeserialize(const void *buffer, size_t length) {
  return new T(buffer, length);
}
REGISTER_TRT_PLUGIN("swish_plugin", PluginTensorRT,
                    CreateSwishPluginDeserialize<SwishPlugin>);

int SwishPlugin::initialize() { return 0; }

nvinfer1::Dims SwishPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

template <typename T>
__device__ T math_exp(T a);

template <>
__device__ half math_exp<half>(half a) {
  return hexp(a);
}

template <>
__device__ float math_exp<float>(float a) {
  return expf(a);
}

template <typename T>
__global__ void swish_kernel(int num, const T *input, T *output, T beta) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
#if __CUDA_ARCH__ >= 350
    output[index] =
        __ldg(input + index) /
        (static_cast<T>(1.0) + math_exp<T>(-beta * __ldg(input + index)));
#else
    output[index] = input[index] /
                    (static_cast<T>(1.0) + math_exp<T>(-beta * input[index]));
#endif
  }
}

size_t SwishPlugin::getSerializationSize() {
  return getBaseSerializationSize() + SerializedSize(beta_);
}

void SwishPlugin::serialize(void *buffer) {
  SerializeValue(&buffer, getPluginType());
  serializeBase(buffer);
  SerializeValue(&buffer, beta_);
}

int SwishPlugin::enqueue(int batch_size, const void *const *inputs,
                         void **outputs, void *workspace, hipStream_t stream) {
  const auto &input_dims = this->getInputDims(0);
  int num = batch_size;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;

  auto type = getDataType();
  if (type == nvinfer1::DataType::kFLOAT) {
    const float *input = reinterpret_cast<const float *>(inputs[0]);
    float *output = reinterpret_cast<float **>(outputs)[0];
    swish_kernel<float><<<blocks, threads, 0, stream>>>(num, input, output,
                                                        beta_);
  } else if (type == nvinfer1::DataType::kHALF) {
    const half *input = reinterpret_cast<const half *>(inputs[0]);
    half *output = reinterpret_cast<half **>(outputs)[0];
    half beta = static_cast<half>(beta_);
    swish_kernel<half><<<blocks, threads, 0, stream>>>(num, input, output,
                                                       beta_);
  } else {
    PADDLE_THROW("The Swish TRT Plugin's input type should be float or half.");
  }
  // input dims is CHW.
  return hipGetLastError() != hipSuccess;
}

bool SwishPlugin::supportsFormat(nvinfer1::DataType type,
                                 nvinfer1::PluginFormat format) const {
  return ((type == nvinfer1::DataType::kFLOAT ||
           type == nvinfer1::DataType::kHALF) &&
          (format == nvinfer1::PluginFormat::kNCHW));
}

// Dynamic Plugin below.
int SwishPluginDynamic::initialize() { return 0; }

size_t SwishPluginDynamic::getSerializationSize() const {
  return getBaseSerializationSize() + SerializedSize(beta_);
}

void SwishPluginDynamic::serialize(void *buffer) const {
  SerializeValue(&buffer, getPluginType());
  serializeBase(buffer);
  SerializeValue(&buffer, beta_);
}

nvinfer1::DimsExprs SwishPluginDynamic::getOutputDimensions(
    int outputIndex, const nvinfer1::DimsExprs *inputs, int nbInputs,
    nvinfer1::IExprBuilder &exprBuilder) {
  return inputs[0];
}

bool SwishPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  return ((in_out[pos].type == nvinfer1::DataType::kFLOAT ||
           in_out[pos].type == nvinfer1::DataType::kHALF) &&
          in_out[pos].format == nvinfer1::PluginFormat::kNCHW);
}

nvinfer1::DataType SwishPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The Swish Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT ||
                     input_types[0] == nvinfer1::DataType::kHALF),
                    true, platform::errors::InvalidArgument(
                              "The input type should be half or float"));
  return input_types[0];
}

int SwishPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                                const nvinfer1::PluginTensorDesc *output_desc,
                                const void *const *inputs, void *const *outputs,
                                void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  size_t num = ProductDim(input_dims);
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    const float *input = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    swish_kernel<float><<<blocks, threads, 0, stream>>>(num, input, output,
                                                        beta_);
  } else if (input_type == nvinfer1::DataType::kHALF) {
    const half *input = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    half beta = static_cast<half>(beta);
    swish_kernel<half><<<blocks, threads, 0, stream>>>(num, input, output,
                                                       beta);
  } else {
    PADDLE_THROW("The Swish TRT Plugin's input type should be float or half.");
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
