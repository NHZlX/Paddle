#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/transpose_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void transpose_kernel(int nthreads, T* in_data, T* out_data,
                                 int ndims, std::vector<int> axis,
                                 std::vector<int> in_offset,
                                 std::vector<int> out_offset) {
  int to_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int from_index = 0;
    int temp = to_index;
    for (size_t i = 0; i < ndims; i++) {
      from_index += (temp / out_offset[i]) * in_offset[axis[i]];
      temp = temp % out_offset[i];
    }
    out_data[to_index] = in_data[from_index];
  }
}

template <typename T>
class TransposeCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "It must use GPUPlace.");
    auto* in = context.Input<framework::Tensor>("X");
    auto* out = context.Output<framework::Tensor>("Out");
    auto* in_data = in->template data<T>();
    auto* out_data = out->template mutable_data<T>(context.GetPlace());
    auto axis = context.op_.GetAttr<std::vector<int>>("axis");
    auto in_dim = in->dims();
    auto out_dim = out->dims();
    auto data_size = product(in_dim);
    size_t ndims = in_dim.size();

    std::vector<int> in_offset(ndims, 1);
    std::vector<int> out_offset(ndims, 1);

    for (int i = ndims - 2; i >= 0; i--) {
      in_offset[i] = in_offset[i + 1] * in_dim[i + 1];
      out_offset[i] = out_offset[i + 1] * out_dim[i + 1];
    }
    int block = 512;
    int grid = (data_size + block - 1) / block;
    transpose_kernel << grid, block >> (data_size, in_data, out_data, ndims,
                                        axis, in_offset, out_offset);
  }
};

template <T>
class TransposeGradCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "It must use GPUPlace.");
    auto* in = context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* out = context.Output<framework::Tensor>(framework::GradVarName("X"));
    auto* in_data = in->template data<T>();
    auto* out_data = out->template mutable_data<T>(context.GetPlace());
    auto axis_temp = context.op_.GetAttr<std::vector<int>>("axis");
    auto in_dim = in->dims();
    auto out_dim = out->dims();
    auto data_size = product(in_dim);
    size_t ndims = in_dim.size();

    std::vector<int> axis(axis_temp);
    std::vector<int> in_offset(ndims, 1);
    std::vector<int> out_offset(ndims, 1);

    for (size_t i = 0; i < axis.size(); i++) {
      axis[axis_temp[i]] = i;
    }

    for (int i = ndims - 2; i >= 0; i--) {
      in_offset[i] = in_offset[i + 1] * in_dim[i + 1];
      out_offset[i] = out_offset[i + 1] * out_dim[i + 1];
    }

    int block = 512;
    int grid = (data_size + block - 1) / block;
    transpose_kernel << grid, block >> (data_size, in_data, out_data, ndims,
                                        axis, in_offset, out_offset);
  }
};

}  // namespace operators
}  // namespace paddle
namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(transpose, ops::TransposeCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(transpose_grad, ops::TransposeGradCUDAKernel<float>);
